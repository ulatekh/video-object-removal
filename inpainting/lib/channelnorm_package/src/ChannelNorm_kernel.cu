#include "hip/hip_runtime.h"
#include <THC/THC.h>

#define real float

/* Not yet sure how to do this properly... */
#define THCRealTensor_size THFloatTensor_size
#define THCRealTensor_stride THFloatTensor_stride

#define CUDA_NUM_THREADS 512 
#define THREADS_PER_BLOCK 64 

#define DIM0(TENSOR) ((TENSOR).x)
#define DIM1(TENSOR) ((TENSOR).y)
#define DIM2(TENSOR) ((TENSOR).z)
#define DIM3(TENSOR) ((TENSOR).w)

#define DIM3_INDEX(TENSOR, xx, yy, zz, ww) ((TENSOR)[((xx) * (TENSOR##_stride.x)) + ((yy) * (TENSOR##_stride.y)) + ((zz) * (TENSOR##_stride.z)) + ((ww) * (TENSOR##_stride.w))])


#ifdef __cplusplus
    extern "C" {
#endif

__global__ void kernel_ChannelNorm_updateOutput(const int n, const float* input1, const long4 input1_size, const long4 input1_stride, float* output, const long4 output_size, const long4 output_stride, int norm_deg) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    int dim_b = DIM0(output_size);
    int dim_c = DIM1(output_size);
    int dim_h = DIM2(output_size);
    int dim_w = DIM3(output_size);
    int dim_chw = dim_c * dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    int i1dim_c = DIM1(input1_size);
    int i1dim_h = DIM2(input1_size);
    int i1dim_w = DIM3(input1_size);
    int i1dim_chw = i1dim_c * i1dim_h * i1dim_w;
    int i1dim_hw  = i1dim_h * i1dim_w;

    float result = 0.0;

    for (int c = 0; c < i1dim_c; ++c) {
        int i1Index = b * i1dim_chw + c * i1dim_hw + y * i1dim_w + x;
        float val = input1[i1Index];
        result += val * val;
    }
    result = sqrt(result);
    output[index] = result;
}


__global__ void kernel_ChannelNorm_backward_input1(const int n, const float* input1, const long4 input1_size, const long4 input1_stride,
    const float* output, const long4 output_size, const long4 output_stride, const float* gradOutput, const long4 gradOutput_size, const long4 gradOutput_stride,
    float* gradInput, const long4 gradInput_size, const long4 gradInput_stride, int norm_deg) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    float val = 0.0;

    int dim_b = DIM0(gradInput_size);
    int dim_c = DIM1(gradInput_size);
    int dim_h = DIM2(gradInput_size);
    int dim_w = DIM3(gradInput_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;


    int outIndex = b * dim_hw + y * dim_w + x;
    val = gradOutput[outIndex] * input1[index] / (output[outIndex]+1e-9);
    gradInput[index] = val;

}

void ChannelNorm_kernel_forward(THCState* state, THCudaTensor* input1, THCudaTensor* output, int norm_deg) {
    int n = 0;
    
    const long4 input1_size = make_long4(THCTensor_(size)(input1, 0), THCTensor_(size)(input1, 1), THCTensor_(size)(input1, 2), THCTensor_(size)(input1, 3));
    const long4 input1_stride = make_long4(THCTensor_(stride)(input1, 0), THCTensor_(stride)(input1, 1), THCTensor_(stride)(input1, 2), THCTensor_(stride)(input1, 3));

    const long4 output_size = make_long4(THCTensor_(size)(output, 0), THCTensor_(size)(output, 1), THCTensor_(size)(output, 2), THCTensor_(size)(output, 3));
    const long4 output_stride = make_long4(THCTensor_(stride)(output, 0), THCTensor_(stride)(output, 1), THCTensor_(stride)(output, 2), THCTensor_(stride)(output, 3));

    n = THCudaTensor_nElement(state, output);
    kernel_ChannelNorm_updateOutput<<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        n, THCudaTensor_data(state, input1), input1_size, input1_stride, THCudaTensor_data(state, output), output_size, output_stride, 
        norm_deg);

    THCudaCheck(hipGetLastError());
}

void ChannelNorm_kernel_backward(THCState* state, THCudaTensor* input1, THCudaTensor* output, THCudaTensor* gradOutput, THCudaTensor* gradInput1, int norm_deg) {
    int n = 0;

    const long4 input1_size = make_long4(THCTensor_(size)(input1, 0), THCTensor_(size)(input1, 1), THCTensor_(size)(input1, 2), THCTensor_(size)(input1, 3));
    const long4 input1_stride = make_long4(THCTensor_(stride)(input1, 0), THCTensor_(stride)(input1, 1), THCTensor_(stride)(input1, 2), THCTensor_(stride)(input1, 3));

    const long4 output_size = make_long4(THCTensor_(size)(output, 0), THCTensor_(size)(output, 1), THCTensor_(size)(output, 2), THCTensor_(size)(output, 3));
    const long4 output_stride = make_long4(THCTensor_(stride)(output, 0), THCTensor_(stride)(output, 1), THCTensor_(stride)(output, 2), THCTensor_(stride)(output, 3));

    const long4 gradOutput_size = make_long4(THCTensor_(size)(gradOutput, 0), THCTensor_(size)(gradOutput, 1), THCTensor_(size)(gradOutput, 2), THCTensor_(size)(gradOutput, 3));
    const long4 gradOutput_stride = make_long4(THCTensor_(stride)(gradOutput, 0), THCTensor_(stride)(gradOutput, 1), THCTensor_(stride)(gradOutput, 2), THCTensor_(stride)(gradOutput, 3));

    const long4 gradInput1_size = make_long4(THCTensor_(size)(gradInput1, 0), THCTensor_(size)(gradInput1, 1), THCTensor_(size)(gradInput1, 2), THCTensor_(size)(gradInput1, 3));
    const long4 gradInput1_stride = make_long4(THCTensor_(stride)(gradInput1, 0), THCTensor_(stride)(gradInput1, 1), THCTensor_(stride)(gradInput1, 2), THCTensor_(stride)(gradInput1, 3));

    n = THCudaTensor_nElement(state, gradInput1);
    kernel_ChannelNorm_backward_input1<<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        n, THCudaTensor_data(state, input1), input1_size, input1_stride, THCudaTensor_data(state, output), output_size, output_stride,
        THCudaTensor_data(state, gradOutput), gradOutput_size, gradOutput_stride, THCudaTensor_data(state, gradInput1), gradInput1_size, gradInput1_stride,
        norm_deg
    );

    THCudaCheck(hipGetLastError());
}

#ifdef __cplusplus
    }
#endif
